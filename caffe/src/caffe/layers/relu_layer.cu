#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

	template <typename Dtype>
		__global__ void ReLUForward(const int n, const Dtype* in, Dtype* out,
				Dtype negative_slope) {
			CUDA_KERNEL_LOOP(index, n) {
				out[index] = in[index] > 0 ? in[index] : in[index] * negative_slope;
			}
		}

	template <typename Dtype>
		void ReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
				const vector<Blob<Dtype>*>& top) {
			const Dtype* bottom_data = bottom[0]->gpu_data();
			Dtype* top_data = top[0]->mutable_gpu_data();
			const int count = bottom[0]->count();
			Dtype negative_slope = this->layer_param_.relu_param().negative_slope();
			// NOLINT_NEXT_LINE(whitespace/operators)
			ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
					count, bottom_data, top_data, negative_slope);
			CUDA_POST_KERNEL_CHECK;
			// << " count: " << count << " bottom_data: "
			//     << (unsigned long)bottom_data
			//     << " top_data: " << (unsigned long)top_data
			//     << " blocks: " << CAFFE_GET_BLOCKS(count)
			//     << " threads: " << CAFFE_CUDA_NUM_THREADS;
		}

	template <typename Dtype>
		__global__ void ReLUBackward(const int n, const Dtype* in_diff,
				const Dtype* in_data, Dtype* out_diff, Dtype negative_slope) {
			CUDA_KERNEL_LOOP(index, n) {
				out_diff[index] = in_diff[index] * ((in_data[index] > 0)
						+ (in_data[index] <= 0) * negative_slope);
			}
		}

	template <typename Dtype>
		void ReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
				const vector<bool>& propagate_down,
				const vector<Blob<Dtype>*>& bottom) {
			if (propagate_down[0]) {
				const Dtype* bottom_data = bottom[0]->gpu_data();
				const Dtype* top_diff = top[0]->gpu_diff();
				Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
				const int count = bottom[0]->count();
				Dtype negative_slope = this->layer_param_.relu_param().negative_slope();
				// NOLINT_NEXT_LINE(whitespace/operators)
				ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
						count, top_diff, bottom_data, bottom_diff, negative_slope);
				CUDA_POST_KERNEL_CHECK;
			}
			//added by ysm 
			//static int print_count = 0;
			//if(print_count%100 == 0) {
			//	if(this->layer_param_.name() == "relu3") {
			//		std::cout << this->layer_param_.name() << std::endl;
			//		const Dtype* topdiff = top[0]->cpu_diff();
			//		const Dtype* bottomdiff = bottom[0]->cpu_diff();
			//		std::cout << "top diff:";
			//		int dim = top[0]->width()*top[0]->height();
			//		for(int i = 0; i < dim; ++i) {
			//			std::cout << topdiff[i] << " ";
			//		}
			//		std::cout << std::endl;
			//		std::cout << "bottom diff:";
			//		for(int i = 0; i < dim; ++i) {
			//			std::cout << bottomdiff[i] << " ";
			//		}
			//		std::cout << std::endl;
			//	}
			//}
			//if(this->layer_param_.name() == "relu3")
			//	print_count++;
			//added by ysm end
		}


	INSTANTIATE_LAYER_GPU_FUNCS(ReLULayer);


}  // namespace caffe
