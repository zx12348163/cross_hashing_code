#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void kernel_channel_max(const int num, const int channels,
    const int spatial_dim, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype maxval = -FLT_MAX;
    for (int c = 0; c < channels; ++c) {
      maxval = max(data[(n * channels + c) * spatial_dim + s], maxval);
    }
    out[index] = maxval;
  }
}

template <typename Dtype>
__global__ void kernel_channel_subtract(const int count,
    const int num, const int channels,
    const int spatial_dim, const Dtype* channel_max, Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / channels / spatial_dim;
    int s = index % spatial_dim;
    data[index] -= channel_max[n * spatial_dim + s];
  }
}

template <typename Dtype>
__global__ void kernel_exp(const int count, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, count) {
    out[index] = exp(data[index]);
  }
}

template <typename Dtype>
__global__ void kernel_channel_sum(const int num, const int channels,
    const int spatial_dim, const Dtype* data, Dtype* channel_sum) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype sum = 0;
    for (int c = 0; c < channels; ++c) {
      sum += data[(n * channels + c) * spatial_dim + s];
    }
    channel_sum[index] = sum;
  }
}

template <typename Dtype>
__global__ void kernel_channel_div(const int count,
    const int num, const int channels,
    const int spatial_dim, const Dtype* channel_sum, Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / channels / spatial_dim;
    int s = index % spatial_dim;
    data[index] /= channel_sum[n * spatial_dim + s];
  }
}

template <typename Dtype>
__global__ void kernel_channel_dot(const int num, const int channels,
    const int spatial_dim, const Dtype* data_1, const Dtype* data_2,
    Dtype* channel_dot) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype dot = 0;
    for (int c = 0; c < channels; ++c) {
      dot += (data_1[(n * channels + c) * spatial_dim + s]
          * data_2[(n * channels + c) * spatial_dim + s]);
    }
    channel_dot[index] = dot;
  }
}

template <typename Dtype>
void SoftmaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
///LOG(INFO)<<"softmax_layer : Forward_gpu";
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* scale_data = scale_.mutable_gpu_data();
  int count = bottom[0]->count();
  int channels = top[0]->shape(softmax_axis_);
  caffe_copy(count, bottom_data, top_data);
  // We need to subtract the max to avoid numerical issues, compute the exp,
  // and then normalize.
  // compute max
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_max<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_, top_data,
      scale_data);
  // subtract
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_subtract<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num_, channels, inner_num_,
      scale_data, top_data);
  // exponentiate
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_exp<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_data, top_data);
  // sum after exp
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_sum<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_, top_data,
      scale_data);
  // divide
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_div<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num_, channels, inner_num_,
      scale_data, top_data);
/**
  for (int i = 0; i < 128; ++ i) {
    std::cout<<i+1<<std::endl;
    for (int j = 0; j < 6; ++ j) {
      for (int k = 0; k < 6; ++ k) {
        std::cout << top[0]->data_at(i,j*6+k,0,0)<<" ";
      }
      std::cout<<std::endl;
    }
    std::cout<<std::endl;
  }
**/
}

template <typename Dtype>
void SoftmaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype* scale_data = scale_.mutable_gpu_data();
  //added by ysm 
  //static int print_count = 0;
  //top_diff = top[0]->cpu_diff();
  //bottom_diff = bottom[0]->mutable_cpu_diff();
  //Dtype sum;
  //Dtype max;
  //Dtype min;

  //if(print_count%100 == 0) {
  //std::cout << this->layer_param_.name() << std::endl;
  //std::cout << "top diff:";
  //max = top_diff[0];
  //min = top_diff[0];
  //sum = 0;
  //for(int i = 0; i < top[0]->count(); ++i) {
  //    if(max < top_diff[i]) max = top_diff[i];
  //    if(min > top_diff[i]) min = top_diff[i];
  //    sum += top_diff[i];
  //}
  //std::cout << "[max:" << max << " min:" << min << " mean:" << sum/top[0]->count() << "]" << std::endl;
  //std::cout << std::endl;
  //}
  //added by ysm end


  int count = top[0]->count();
  int channels = top[0]->shape(softmax_axis_);
  caffe_copy(count, top_diff, bottom_diff);
  // Compute inner1d(top_diff, top_data) and subtract them from the bottom diff.
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_dot<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_,
      top_diff, top_data, scale_data);
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_subtract<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num_, channels, inner_num_,
      scale_data, bottom_diff);
  // elementwise multiplication
  caffe_gpu_mul<Dtype>(top[0]->count(), bottom_diff, top_data, bottom_diff);
  
  
  //add by ysm
  //if(print_count%100 == 0) {
  //std::cout << "bottom diff:";
  //sum = 0;
  //max = bottom_diff[0];
  //min = bottom_diff[0];
  //for(int i = 0; i < bottom[0]->count(); ++i) {
  //    if(max < bottom_diff[i]) max = bottom_diff[i];
  //    if(min > bottom_diff[i]) min = bottom_diff[i];
  //    sum += bottom_diff[i];
  //}
  //std::cout << "[max:" << max << " min:" << min << " mean:" << sum/bottom[0]->count() << "]" << std::endl;
  //std::cout << std::endl;
  //}
  //print_count++;
  //added by ysm end
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxLayer);


}  // namespace caffe
