#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Slice(const int nthreads, const Dtype* in_data,
    const bool forward, const int num_slices, const int slice_size,
    const int bottom_slice_axis, const int top_slice_axis,
    const int offset_slice_axis, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int total_slice_size = slice_size * top_slice_axis;
    const int slice_num = index / total_slice_size;
    const int slice_index = index % total_slice_size;
    const int bottom_index = slice_index +
        (slice_num * bottom_slice_axis + offset_slice_axis) * slice_size;
    if (forward) {
      out_data[index] = in_data[bottom_index];
    } else {
      out_data[bottom_index] = in_data[index];
    }
  }
}

template <typename Dtype>
void SliceLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  if (top.size() == 1) { return; }
  int offset_slice_axis = 0;
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const int bottom_slice_axis = bottom[0]->shape(slice_axis_);
  const bool kForward = true;
  for (int i = 0; i < top.size(); ++i) {
    Dtype* top_data = top[i]->mutable_gpu_data();
    const int top_slice_axis = top[i]->shape(slice_axis_);
    const int top_slice_size = top_slice_axis * slice_size_;
    const int nthreads = top_slice_size * num_slices_;
    Slice<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom_data, kForward, num_slices_, slice_size_,
        bottom_slice_axis, top_slice_axis, offset_slice_axis, top_data);
    offset_slice_axis += top_slice_axis;
  }
}

template <typename Dtype>
void SliceLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  //Backward_cpu(top, propagate_down, bottom);
  if (!propagate_down[0] || top.size() == 1) { return; }
  int offset_slice_axis = 0;
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int bottom_slice_axis = bottom[0]->shape(slice_axis_);
  const bool kForward = false;
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    const int top_slice_axis = top[i]->shape(slice_axis_);
    const int top_slice_size = top_slice_axis * slice_size_;
    const int nthreads = top_slice_size * num_slices_;
    Slice<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, top_diff, kForward, num_slices_, slice_size_,
        bottom_slice_axis, top_slice_axis, offset_slice_axis, bottom_diff);
    offset_slice_axis += top_slice_axis;
  }
  //added by ysm 
  //static int print_count = 0;
  //if(print_count%100 == 0) {
  //const Dtype* bottom_diff = bottom[0]->cpu_diff();
  //const Dtype* weight_diff = top[0]->cpu_diff();
  //const Dtype* map_diff = top[1]->cpu_diff();
  //Dtype sum;
  //Dtype max;
  //Dtype min;

  //std::cout << this->layer_param_.name() << std::endl;
  //std::cout << "bottom diff:";
  //max = bottom_diff[0];
  //min = bottom_diff[0];
  //sum = 0;
  //for(int i = 0; i < bottom[0]->count(); ++i) {
  //    if(max < bottom_diff[i]) max = bottom_diff[i];
  //    if(min > bottom_diff[i]) min = bottom_diff[i];
  //    sum += bottom_diff[i];
  //}
  //std::cout << "[max:" << max << " min:" << min << " mean:" << sum/bottom[0]->count() << "]" << std::endl;
  //std::cout << std::endl;

  //std::cout << "top[0] diff:";
  //max = weight_diff[0];
  //min = weight_diff[0];
  //sum = 0;
  //for(int i = 0; i < top[0]->count(); ++i) {
  //    if(max < weight_diff[i]) max = weight_diff[i];
  //    if(min > weight_diff[i]) min = weight_diff[i];
  //    sum += weight_diff[i];
  //}
  //std::cout << "[max:" << max << " min:" << min << " mean:" << sum/top[0]->count() << "]" << std::endl;
  //std::cout << std::endl;
  //
  //std::cout << "top[1] diff:";
  //max = map_diff[0];
  //min = map_diff[0];
  //sum = 0;
  //for(int i = 0; i < top[1]->count(); ++i) {
  //    if(max < map_diff[i]) max = map_diff[i];
  //    if(min > map_diff[i]) min = map_diff[i];
  //    sum += map_diff[i];
  //}
  //std::cout << "[max:" << max << " min:" << min << " mean:" << sum/top[1]->count() << "]" << std::endl;
  //std::cout << std::endl;
  //}
  //print_count++;
  //added by ysm end
}

INSTANTIATE_LAYER_GPU_FUNCS(SliceLayer);

}  // namespace caffe
