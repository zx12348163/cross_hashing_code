#include "hip/hip_runtime.h"
// Copyright 2013 Yangqing Jia

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;
using std::min;

namespace caffe {


  template <typename Dtype>
  __global__ void AveForwardLayer(const int nthread, const int n_proposal, const int label_num, const int patch_dim, const Dtype* bottom_data, const Dtype* score_mat, Dtype* top_data) {
    CUDA_KERNEL_LOOP(index, nthread) {
      int d = index % patch_dim;
      int la = (index / patch_dim) % label_num;
      int p = (index / patch_dim / label_num) % n_proposal;
      top_data[la*patch_dim + d] += score_mat[p*label_num + la] * bottom_data[p*patch_dim + d]/n_proposal;
    }
  }


  template <typename Dtype>
  __global__ void AveBackwardLayer(const int nthread, const int n_proposal, const int label_num, const int patch_dim, const Dtype* top_diff, const Dtype* score_mat, Dtype* bottom_diff) {
    CUDA_KERNEL_LOOP(index, nthread) {
      int p = index % n_proposal;
      int la = (index / n_proposal) % label_num;
      int d = (index / n_proposal / label_num) % patch_dim;
      bottom_diff[p*patch_dim + d] += score_mat[p*label_num + la] * top_diff[la*patch_dim + d]/n_proposal;
    }
  }



  // Forward_cpu for FusionLayer
  template <typename Dtype>
  void FusionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, 
					   const vector<Blob<Dtype>*>& top){
    // const Dtype* bottom_data = bottom[0]->gpu_data();
    // Dtype* top_data = (top)[0]->mutable_cpu_data();
    // //Different patches are fusing into one image via max_pooling
    // const Dtype* score_mat = bottom[1]->gpu_data();
    // const Dtype* conv5_scales = bottom[2]->cpu_data();
    // const int n_scales = bottom[2]->channels();
    // caffe_set(top[0]->count(), Dtype(0), top_data);
    // top_data = (top)[0]->mutable_gpu_data();
   
    // for (int n = 0; n < img_num_; ++n){
    //   int n_proposal = conv5_scales[n*n_scales];
    //   int nthread = n_proposal * label_num_ * patch_dim_;
    //   AveForwardLayer<<<CAFFE_GET_BLOCKS(nthread), CAFFE_CUDA_NUM_THREADS>>>(
    // 									     nthread, n_proposal, label_num_, patch_dim_, bottom_data,score_mat, top_data);

    //   bottom_data += bottom[0]->offset(patch_num_each_img_);
    //   score_mat +=  bottom[1]->offset(patch_num_each_img_);
    //   top_data += (top)[0]->offset(1);
    // }
     Forward_cpu(bottom, top);
  }
  // Backward_cpu for FusionLayer
  template <typename Dtype>
  void FusionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, 
					    const vector<bool>& propagate_down,const vector<Blob<Dtype>*>& bottom){
    Backward_cpu(top, propagate_down, bottom);
    // if (!propagate_down[0]){
    //   //return Dtype(0.);
    // }
    // const Dtype* top_diff = top[0]->gpu_diff();
    // Dtype* bottom_diff = (bottom)[0]->mutable_cpu_diff();
    // const Dtype* score_mat = bottom[1]->gpu_data();
    // memset(bottom_diff, 0, (bottom)[0]->count() * sizeof(Dtype));
    // bottom_diff = (bottom)[0]->mutable_gpu_diff();
    // const Dtype* conv5_scales = bottom[2]->cpu_data();
    // const int n_scales = bottom[2]->channels();
    // for (int n = 0; n < img_num_; ++n){
    //   int n_proposal = conv5_scales[n*n_scales];
    //   int nthread = n_proposal * label_num_ * patch_dim_;
    //   AveBackwardLayer<<<CAFFE_GET_BLOCKS(nthread), CAFFE_CUDA_NUM_THREADS>>>(
    // 									      nthread, n_proposal, label_num_, patch_dim_,top_diff, score_mat, bottom_diff);
    //   score_mat += (bottom)[1]->offset(patch_num_each_img_);
    //   bottom_diff += (bottom)[0]->offset(patch_num_each_img_);
    //   top_diff += top[0]->offset(1);
    // }
  }
  
  INSTANTIATE_LAYER_GPU_FUNCS(FusionLayer);
}  // namespace caffe
