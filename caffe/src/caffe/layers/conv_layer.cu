#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

	template <typename Dtype>
		void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
				const vector<Blob<Dtype>*>& top) {
			//LOG(INFO)<<"convolution_layer:Forward_gpu";
			const Dtype* weight = this->blobs_[0]->gpu_data();
			for (int i = 0; i < bottom.size(); ++i) {
				const Dtype* bottom_data = bottom[i]->gpu_data();
				Dtype* top_data = top[i]->mutable_gpu_data();
				for (int n = 0; n < this->num_; ++n) {
					this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
							top_data + n * this->top_dim_);
					if (this->bias_term_) {
						const Dtype* bias = this->blobs_[1]->gpu_data();
						this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
					}
				}
			}

		}

	template <typename Dtype>
		void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
				const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
			const Dtype* weight = this->blobs_[0]->gpu_data();
			Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
			for (int i = 0; i < top.size(); ++i) {
				const Dtype* top_diff = top[i]->gpu_diff();
				// Bias gradient, if necessary.
				if (this->bias_term_ && this->param_propagate_down_[1]) {
					Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
					for (int n = 0; n < this->num_; ++n) {
						this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
					}
				}
				if (this->param_propagate_down_[0] || propagate_down[i]) {
					const Dtype* bottom_data = bottom[i]->gpu_data();
					Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
					for (int n = 0; n < this->num_; ++n) {
						// gradient w.r.t. weight. Note that we will accumulate diffs.
						if (this->param_propagate_down_[0]) {
							this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
									top_diff + n * this->top_dim_, weight_diff);
						}
						// gradient w.r.t. bottom data, if necessary.
						if (propagate_down[i]) {
							this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
									bottom_diff + n * this->bottom_dim_);
						}
					}
				}
			}
			//added by ysm
			//static int print_count = 0;
			//if(print_count%100 == 0) {
			//	if(this->layer_param_.name() == "weight_cccp5"||this->layer_param_.name() == "weight_cccp3"){
			//		std::cout << this->layer_param_.name() << std::endl;
			//		Dtype sum;
			//		Dtype max;
			//		Dtype min;
			//		int top_dim = top[0]->width()*top[0]->height();
			//		int bottom_dim = bottom[0]->width()*bottom[0]->height();
			//		int weight_dim = this->blobs_[0]->count()/this->blobs_[0]->num();
			//		
			//		std::cout << "topdata:";
			//		const Dtype* topdata = top[0]->cpu_data();
			//		sum = 0;
			//		max = topdata[0];
			//		min = topdata[0];
			//		for(int i = 0; i < top[0]->count(); ++i) {
			//			//std::cout << topdata[i] << " ";
			//			if(max < topdata[i]) max = topdata[i];
			//			if(min > topdata[i]) min = topdata[i];
			//			sum += topdata[i];
			//		}
			//		std::cout << "max:" << max << " min:" << min << " mean:" << sum/top[0]->count() << std::endl;
			//		std::cout << std::endl;
			//		
			//		std::cout << "bottomdata:";
			//		const Dtype* bottomdata = bottom[0]->cpu_data();
			//		sum = 0;
			//		max = bottomdata[0];
			//		min = bottomdata[0];
			//		for(int i = 0; i < bottom[0]->count(); ++i) {
			//			//std::cout << bottomdata[i] << " ";
			//			if(max < bottomdata[i]) max = bottomdata[i];
			//			if(min > bottomdata[i]) min = bottomdata[i];
			//			sum += bottomdata[i];
			//		}
			//		std::cout << "max:" << max << " min:" << min << " mean:" << sum/bottom[0]->count() << std::endl;
			//		std::cout << std::endl;
			//		
			//		std::cout << "topdiff:";
			//		const Dtype* topdiff = top[0]->cpu_diff();
			//		sum = 0;
			//		max = topdiff[0];
			//		min = topdiff[0];
			//		for(int i = 0; i < top[0]->count(); ++i) {
			//			//std::cout << topdiff[i] << " ";
			//			if(max < topdiff[i]) max = topdiff[i];
			//			if(min > topdiff[i]) min = topdiff[i];
			//			sum += topdiff[i];
			//		}
			//		std::cout << "max:" << max << " min:" << min << " mean:" << sum/top[0]->count() << std::endl;
			//		std::cout << std::endl;

			//		std::cout << "bottomdiff:";
			//		const Dtype* bottomdiff = bottom[0]->cpu_diff();
			//		sum = 0;
			//		max = bottomdiff[0];
			//		min = bottomdiff[0];
			//		for(int i = 0; i < bottom[0]->count(); ++i) {
			//			//std::cout << bottomdiff[i] << " ";
			//			if(max < bottomdiff[i]) max = bottomdiff[i];
			//			if(min > bottomdiff[i]) min = bottomdiff[i];
			//			sum += bottomdiff[i];
			//		}
			//		std::cout << "max:" << max << " min:" << min << " mean:" << sum/bottom[0]->count() << std::endl;
			//		std::cout << std::endl;

			//		std::cout << "weightdiff:";
			//		const Dtype* weightdiff = this->blobs_[0]->cpu_diff();
			//		sum = 0;
			//		max = weightdiff[0];
			//		min = weightdiff[0];
			//		for(int i = 0; i < this->blobs_[0]->count(); ++i) {
			//			//std::cout << weightdiff[i] << " ";
			//			if(max < weightdiff[i]) max = weightdiff[i];
			//			if(min > weightdiff[i]) min = weightdiff[i];
			//			sum += weightdiff[i];
			//		}
			//		std::cout << "max:" << max << " min:" << min << " mean:" << sum/this->blobs_[0]->count() << std::endl;
			//		std::cout << std::endl;
			//		
			//		std::cout << "weight:";
			//		const Dtype* weightdata = this->blobs_[0]->cpu_data();
			//		sum = 0;
			//		max = weightdata[0];
			//		min = weightdata[0];
			//		for(int i = 0; i < this->blobs_[0]->count(); ++i) {
			//			//std::cout << weightdiff[i] << " ";
			//			if(max < weightdata[i]) max = weightdata[i];
			//			if(min > weightdata[i]) min = weightdata[i];
			//			sum += weightdata[i];
			//		}
			//		std::cout << "max:" << max << " min:" << min << " mean:" << sum/this->blobs_[0]->count() << std::endl;
			//		std::cout << std::endl;
			//	}
			//}
			//if(this->layer_param_.name() == "weight_conv1")
			//	print_count++;
			//added by ysm end
		}

	INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
